#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"


__global__ void CUDA_ColorConversion(unsigned char* src, unsigned char* dst,
                                int rows, int cols, int channels) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  if (col < cols && row < rows) {
    int rgb_offset = (row * cols + col) * channels;
    dst[rgb_offset + 0] = src[rgb_offset + 2];
    dst[rgb_offset + 1] = src[rgb_offset + 1];
    dst[rgb_offset + 2] = src[rgb_offset + 0];
  }
}

__global__ void CUDA_ColorConversion(PtrStep<byte> src, PtrStep<byte> dst,
                                     int rows,
                                int cols, int step, int channels) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;

  if (col < cols && row < rows) {
    int rgb_offset = (row * step + channels * col);
    dst[rgb_offset + 0] = src[rgb_offset + 2];
    dst[rgb_offset + 1] = src[rgb_offset + 1];
    dst[rgb_offset + 2] = src[rgb_offset + 0];
  }
}

__global__ void CUDA_PixelShifting(PtrStep<byte> src, PtrStep<byte> depth,
                                   PtrStep<byte> dst, int rows, int cols,
                                   int step, int channels) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;

  if (col < cols && row < rows) {
    int rgb_offset = (row * step + channels * col);
    int depth_offset = (row * depth.step +  col);

    int dis = (int)((1.0 + depth[depth_offset]) * 5);
    if (col > dis) {
      dst[rgb_offset + 0 - dis * channels] = src[rgb_offset + 0];
      dst[rgb_offset + 1 - dis * channels] = src[rgb_offset + 1];
      dst[rgb_offset + 2 - dis * channels] = src[rgb_offset + 2];
    }
  }
}

void BGR2RGB(unsigned char* h_image, int height, int width, int channels) {
  unsigned char* d_bgr_image = NULL;
  unsigned char* d_rgb_image = NULL;

  // allocate the memory in gpu
  hipMalloc((void**)&d_bgr_image, height * width * channels);
  hipMalloc((void**)&d_rgb_image, height * width * channels);

  // copy data from CPU to GPU
  hipMemcpy(d_bgr_image, h_image, height * width * channels,
             hipMemcpyHostToDevice);

  const dim3 dimGrid((int)ceil(width / 16.), (int)ceil(height / 16.));
  const dim3 dimBlock(16, 16);
  CUDA_ColorConversion<<<dimGrid, dimBlock>>>(d_bgr_image, d_rgb_image, height,
                                         width, channels);

  // copy processed data back to cpu from gpu
  hipMemcpy(h_image, d_rgb_image, height * width * channels,
             hipMemcpyDeviceToHost);

  // free gpu mempry
  hipFree(d_bgr_image);
  hipFree(d_rgb_image);
}

void BGR2RGB(PtrStep<byte> src, PtrStep<byte> dst, int height, int width,
             int step, int channels) {
  const dim3 dimGrid((int)ceil(width / 16.), (int)ceil(height / 16.));
  const dim3 dimBlock(16, 16);
  CUDA_ColorConversion<<<dimGrid, dimBlock>>>(src, dst, height, width, step,
                                         channels);
}

void PixelShifting(PtrStep<byte> src, PtrStep<byte> depth, PtrStep<byte> dst,
                   int height, int width, int step, int channels) {
  const dim3 dimGrid((int)ceil(width / 16.), (int)ceil(height / 16.));
  const dim3 dimBlock(16, 16);
  CUDA_PixelShifting<<<dimGrid, dimBlock>>>(src, depth, dst, height, width,
                                            step, channels);
}


