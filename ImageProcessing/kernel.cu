#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"

__global__ void CUDA_ColorConversion(unsigned char* src, unsigned char* dst,
                                     int rows, int cols, int channels) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  if (col < cols && row < rows) {
    int rgb_offset = (row * cols + col) * channels;
    dst[rgb_offset + 0] = src[rgb_offset + 2];
    dst[rgb_offset + 1] = src[rgb_offset + 1];
    dst[rgb_offset + 2] = src[rgb_offset + 0];
  }
}

__global__ void CUDA_ColorConversion(PtrStep<byte> src, PtrStep<byte> dst,
                                     int rows, int cols, int step,
                                     int channels) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;

  if (col < cols && row < rows) {
    int rgb_offset = (row * step + channels * col);
    dst[rgb_offset + 0] = src[rgb_offset + 2];
    dst[rgb_offset + 1] = src[rgb_offset + 1];
    dst[rgb_offset + 2] = src[rgb_offset + 0];
  }
}

__global__ void CUDA_PixelShifting(PtrStep<byte> src, PtrStep<byte> depth,
                                   PtrStep<byte> dst, int rows, int cols,
                                   int step, int channels) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;

  if (col < cols && row < rows) {
    int rgb_offset = (row * step + channels * col);
    int depth_offset = (row * depth.step + col);

    int dis = (int)(depth[depth_offset] * 30 / 255);
    if (col > dis) {
      dst[rgb_offset + 0 - dis * channels] = src[rgb_offset + 0];
      dst[rgb_offset + 1 - dis * channels] = src[rgb_offset + 1];
      dst[rgb_offset + 2 - dis * channels] = src[rgb_offset + 2];
    }
  }
}

__global__ void CUDA_ImagePainting(PtrStep<byte> img,
                                   int rows, int cols, int step, int channels) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;

  if (col < cols && row < rows) {
    int rgb_offset = (row * step + channels * col);
    if (img[rgb_offset + 0] == 0 && img[rgb_offset + 1] == 0 &&
        img[rgb_offset + 2] == 0) {
      for (int offset = 1; offset < 30; offset++) {
        if (col + offset <= cols &&
            img[rgb_offset + 0 + offset * channels] != 0 ||
            img[rgb_offset + 1 + offset * channels] != 0 ||
            img[rgb_offset + 2 + offset * channels] != 0) {
          img[rgb_offset + 0] = img[rgb_offset + 0 + offset * channels];
          img[rgb_offset + 1] = img[rgb_offset + 1 + offset * channels];
          img[rgb_offset + 2] = img[rgb_offset + 2 + offset * channels];
          break;
        }
        if (col + offset <= cols &&
            img[rgb_offset + 0 - offset * channels] != 0 ||
            img[rgb_offset + 1 - offset * channels] != 0 ||
            img[rgb_offset + 2 - offset * channels] != 0) {
          img[rgb_offset + 0] = img[rgb_offset + 0 - offset * channels];
          img[rgb_offset + 1] = img[rgb_offset + 1 - offset * channels];
          img[rgb_offset + 2] = img[rgb_offset + 2 - offset * channels];
          break;
        }
      }
    }
  }
}

void BGR2RGB(unsigned char* h_image, int height, int width, int channels) {
  unsigned char* d_bgr_image = NULL;
  unsigned char* d_rgb_image = NULL;

  // allocate the memory in gpu
  hipMalloc((void**)&d_bgr_image, height * width * channels);
  hipMalloc((void**)&d_rgb_image, height * width * channels);

  // copy data from CPU to GPU
  hipMemcpy(d_bgr_image, h_image, height * width * channels,
             hipMemcpyHostToDevice);

  const dim3 dimGrid((int)ceil(width / 16.), (int)ceil(height / 16.));
  const dim3 dimBlock(16, 16);
  CUDA_ColorConversion<<<dimGrid, dimBlock>>>(d_bgr_image, d_rgb_image, height,
                                              width, channels);

  // copy processed data back to cpu from gpu
  hipMemcpy(h_image, d_rgb_image, height * width * channels,
             hipMemcpyDeviceToHost);

  // free gpu mempry
  hipFree(d_bgr_image);
  hipFree(d_rgb_image);
}

void BGR2RGB(PtrStep<byte> src, PtrStep<byte> dst, int height, int width,
             int step, int channels) {
  const dim3 dimGrid((int)ceil(width / 16.), (int)ceil(height / 16.));
  const dim3 dimBlock(16, 16);
  CUDA_ColorConversion<<<dimGrid, dimBlock>>>(src, dst, height, width, step,
                                              channels);
}

void PixelShifting(PtrStep<byte> src, PtrStep<byte> depth, PtrStep<byte> dst,
                   int height, int width, int step, int channels) {
  const dim3 dimGrid((int)ceil(width / 16.), (int)ceil(height / 16.));
  const dim3 dimBlock(16, 16);
  CUDA_PixelShifting<<<dimGrid, dimBlock>>>(src, depth, dst, height, width,
                                            step, channels);
}

void ImagePainting(PtrStep<byte> img, int height, int width,
                   int step, int channels) {
  const dim3 dimGrid((int)ceil(width / 16.), (int)ceil(height / 16.));
  const dim3 dimBlock(16, 16);
  CUDA_ImagePainting<<<dimGrid, dimBlock>>>(img, height, width, step, channels);
}
