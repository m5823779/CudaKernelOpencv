#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"
#include <iostream>

__global__ void Conversion_CUDA(unsigned char* d_bgr_image, unsigned char* d_rgb_image, int rows, int cols, int channels)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	if (col < cols && row < rows)
	{
		int rgb_offset = (row * cols + col) * channels;
		d_rgb_image[rgb_offset + 0] = d_bgr_image[rgb_offset + 2];
		d_rgb_image[rgb_offset + 1] = d_bgr_image[rgb_offset + 1];
		d_rgb_image[rgb_offset + 2] = d_bgr_image[rgb_offset + 0];

	}
}

void ColorBGR2RGB(unsigned char* h_image, int height, int width, int channels) 
{
	unsigned char* d_bgr_image = NULL;
	unsigned char* d_rgb_image = NULL;

	//allocate the memory in gpu
	hipMalloc((void**)&d_bgr_image, height * width * channels);
	hipMalloc((void**)&d_rgb_image, height * width * channels);


	//copy data from CPU to GPU
	hipMemcpy(d_bgr_image, h_image, height * width * channels, hipMemcpyHostToDevice);

	const dim3 dimGrid((int)ceil((width) / 16), (int)ceil((height) / 16));
	const dim3 dimBlock(16, 16);
	Conversion_CUDA <<<dimGrid, dimBlock>>> (d_bgr_image, d_rgb_image, height, width, channels);

	//copy processed data back to cpu from gpu
	hipMemcpy(h_image, d_rgb_image, height * width * channels, hipMemcpyDeviceToHost);

	//free gpu mempry
	hipFree(d_bgr_image);
	hipFree(d_rgb_image);
}

void ColorBGR2RGB_GpuMat(unsigned char* h_image, int height, int width, int channels)
{
	unsigned char* d_bgr_image = NULL;
	unsigned char* d_rgb_image = NULL;

	//allocate the memory in gpu
	hipMalloc((void**)&d_bgr_image, height * width * channels);
	hipMalloc((void**)&d_rgb_image, height * width * channels);


	//copy data from CPU to GPU
	hipMemcpy(d_bgr_image, h_image, height * width * channels, hipMemcpyDeviceToDevice);

	const dim3 dimGrid((int)ceil((width) / 16), (int)ceil((height) / 16));
	const dim3 dimBlock(16, 16);
	Conversion_CUDA << <dimGrid, dimBlock >> > (d_bgr_image, d_rgb_image, height, width, channels);

	//copy processed data back to cpu from gpu
	hipMemcpy(h_image, d_rgb_image, height * width * channels, hipMemcpyDeviceToDevice);

	//free gpu mempry
	hipFree(d_bgr_image);
	hipFree(d_rgb_image);
}
